#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelCount = width * height;

    if (tid < pixelCount) {
        int pixelIndex = tid * 4;

        // Invert R, G, B components, leave A unchanged
        for (int i = 0; i < 3; ++i) {
            image[pixelIndex + i] = 255 - image[pixelIndex + i];
        }
    }
}

void solve(unsigned char* image, int width, int height) {
    unsigned char* d_image;
    int image_size = width * height * 4;

    // Allocate device memory
    hipMalloc(&d_image, image_size * sizeof(unsigned char));

    // Copy input data from host to device
    hipMemcpy(d_image, image, image_size * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, width, height);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(image, d_image, image_size * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_image);
}
