#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < input_size + kernel_size){
        for (int j = 0; j < kernel_size; j += 1){
            output[tid] += input[tid+j] * kernel[j];
            }
    }
}

void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    float *d_input, *d_kernel, *d_output;
    int output_size = input_size - kernel_size + 1;

    // Allocate device memory
    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_kernel, kernel_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_kernel, d_output, input_size, kernel_size);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}
